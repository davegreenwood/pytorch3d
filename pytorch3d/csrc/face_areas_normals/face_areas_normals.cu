#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <ATen/ATen.h>
#include <tuple>

template <typename scalar_t>
__global__ void FaceAreasNormalsKernel(
    const scalar_t* __restrict__ verts,
    const long* __restrict__ faces,
    scalar_t* __restrict__ face_areas,
    scalar_t* __restrict__ face_normals,
    const size_t V,
    const size_t F) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;

  // Faces split evenly over the number of threads in the grid.
  // Each thread computes the area & normal of its respective faces and adds it
  // to the global face_areas tensor.
  for (size_t f = tid; f < F; f += stride) {
    const long i0 = faces[3 * f + 0];
    const long i1 = faces[3 * f + 1];
    const long i2 = faces[3 * f + 2];

    const scalar_t v0_x = verts[3 * i0 + 0];
    const scalar_t v0_y = verts[3 * i0 + 1];
    const scalar_t v0_z = verts[3 * i0 + 2];

    const scalar_t v1_x = verts[3 * i1 + 0];
    const scalar_t v1_y = verts[3 * i1 + 1];
    const scalar_t v1_z = verts[3 * i1 + 2];

    const scalar_t v2_x = verts[3 * i2 + 0];
    const scalar_t v2_y = verts[3 * i2 + 1];
    const scalar_t v2_z = verts[3 * i2 + 2];

    const scalar_t ax = v1_x - v0_x;
    const scalar_t ay = v1_y - v0_y;
    const scalar_t az = v1_z - v0_z;

    const scalar_t bx = v2_x - v0_x;
    const scalar_t by = v2_y - v0_y;
    const scalar_t bz = v2_z - v0_z;

    const scalar_t cx = ay * bz - az * by;
    const scalar_t cy = az * bx - ax * bz;
    const scalar_t cz = ax * by - ay * bx;

    scalar_t norm = sqrt(cx * cx + cy * cy + cz * cz);
    face_areas[f] = norm / 2.0;
    norm = (norm < 1e-6) ? 1e-6 : norm; // max(norm, 1e-6)
    face_normals[3 * f + 0] = cx / norm;
    face_normals[3 * f + 1] = cy / norm;
    face_normals[3 * f + 2] = cz / norm;
  }
}

std::tuple<at::Tensor, at::Tensor> FaceAreasNormalsCuda(
    at::Tensor verts,
    at::Tensor faces) {
  const auto V = verts.size(0);
  const auto F = faces.size(0);

  at::Tensor areas = at::empty({F}, verts.options());
  at::Tensor normals = at::empty({F, 3}, verts.options());

  const int blocks = 64;
  const int threads = 512;
  AT_DISPATCH_FLOATING_TYPES(verts.type(), "face_areas_normals_cuda", ([&] {
                               FaceAreasNormalsKernel<scalar_t>
                                   <<<blocks, threads>>>(
                                       verts.data_ptr<scalar_t>(),
                                       faces.data_ptr<long>(),
                                       areas.data_ptr<scalar_t>(),
                                       normals.data_ptr<scalar_t>(),
                                       V,
                                       F);
                             }));

  return std::make_tuple(areas, normals);
}
